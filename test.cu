#include <string>
#include <hip/hip_runtime.h>

#include "include/analysis.h"
#include "include/mesh.h"
#include "include/physics.h"
#include "include/tetrahedral.h"

int main(int argc, char *argv[])
{
    using T = double;
    using Basis = TetrahedralBasis;
    using Quadrature = TetrahedralQuadrature;
    using Physics = NeohookeanPhysics<T>;
    using Analysis = FEAnalysis<T, Basis, Quadrature, Physics>;

    std::vector<std::string> node_set_names;

    // Load in the mesh
    std::string filename("../input/Tensile1.inp");
    Mesh<T> tensile;
    tensile.load_mesh(filename);

    // Set the number of degrees of freedom
    int ndof = 3 * tensile.num_nodes;
    for (int i = 0; i < tensile.num_fixed_nodes; i++)
    {
        std::cout << tensile.fixed_nodes[i] << std::endl;
    }
}